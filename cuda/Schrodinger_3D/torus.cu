#include "hip/hip_runtime.h"
#include "const.h"
#include "complex.cu"

// Definizione della struttura torus
//La teoria la trovate nell'articolo "Schrödinger’s Smoke" di Chern, Knoppel et al.

struct Torus {
    int resx, resy, resz; //resolutions (number of samples) in each direction
    int sizex, sizey, sizez; // size of the domain in each direction
    double dx, dy, dz; // edge length of the grid cell

    double *vx;
    double *vy;
    double *vz;

    int plen;
    int yzlen;

    double *div;
    hipDoubleComplex* fftbuf;
    hipfftHandle fftplan;

    double* poissonbuf;
};

__constant__ Torus torus;
Torus torus_cpu;


void torus_ds(Torus* t)
{
    t -> dx = ((double)t -> sizex) /(t -> resx);
    t -> dy = ((double)t -> sizey) /(t->resy);
    t -> dz = ((double)t -> sizez) /(t->resz);
}

__device__ __inline__ int index3d(int i, int j, int k)
{
    return (k +j*torus.resz +i*torus.yzlen);
}

__device__ __inline__ void getCoords(int i, int *x, int *y, int *z)
{
    *x = i/(torus.yzlen);
    int t = i% torus.yzlen;
    *y = t/torus.resz;
    *z = t% torus.resz;
}

__device__ inline__ int check_limit(int limit){

    int i = blockIdx.x*blockdim.x+threadIdx.x;
    if (i< limit)
        return i;
    return -1;
}


__global__ void torus_div()
{
    int normal_index = check_limit(torus.plen);
    if(normal_index <0) return;

    double dx2 = torus.dx *torus.dx;
    double dy2 = torus.dy*torus.dy;
    double dz2 = torus.dz*torus.dz;

    double* vx = torus.vx;
    double* vy = torus.vy;
    double* vz = torus.vz;

    int i, j, k;
    getCoords(normal_index, &i, &j, &k);

    int ixm = (i-1+torus.resx)%torus.resx;
    int iym = (j-1+torus.resy)%torus.resy;
    int izm = (k -1+torus.resz)%torus.resz;

    torus.div[normal_index] = (vx[normal_index] -vx[index3d(ixm, j, k)])/dx2;
    torus.div[normal_index] += (vy[normal_index] -vy[index3d(i, iym, k)])/dy2;
    torus.div[normal_index] += (vz[normal_index]-vz[index3d(i, j, izm)])/dz2;

}

__global__ void torus_div2buf()
{
    int ind = check_limit(torus.plen);
    if (ind<0) return;

    //make_hipDoubleComplex(real part, imaginary part) è una funzione nativa di CUDA
    torus.fftbuf[ind] = make_hipDoubleComplex(torus.div[ind], 0.0);
}

__global__ void torus_Poisson_kernel()
//equazione (18) appendice E sugli autovalori discreti dell'operatore di Laplace
{
    int ind = check_limit(torus.plen);
    if(ind<0) return;
    int i, j, k;
    getCoords(ind, &i, &j, &k);
    double sx = sin(pi*i/torus.resx)/torus.dx;
    double sy = sin(pi*j/torus.resy)/torus.dy;
    double sz = sin(pi*k/torus.resz)/torus.dz;

    double d = sx*sx + sy*sy + sz*sz;
    double factor =0.0f;
    if(ind > 0)
    {
        factor = -0.25/d;
    }
}

int numblock(int limit){
    return(limit+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK
}

void torus_poisson_exe()
{
    int nb = numblock(torus_cpu.plen)
    torus_Poisson_kernel<<<nb, THREADS_PER_BLOCK
    hipDeviceSynchronize();
}
void fftcuda(hipfftDoubleComplex *data)
{
    hipfftExecZ2Z(torus_cpu.fftplan, data, data, HIPFFT_FORWARD);
    hipDeviceSynchronize();
}

void ifftcuda(hipfftDoubleComplex *data)
{
    hipfftExecZ2Z(torus_cpu.fftplan, data, data, HIPFFT_BACKWARD);
    hipDeviceSynchronize();
}
__global__ void PoissonMain()
{
    int ind = check_limit(torus.plen);
    if(ind<0) return;
    complexProd(&torus.fftbuf[ind], torus.poissonbuf[ind]);

}

void Torus_Poisson_solver()
{
    int nb = numblock(torus_cpu.plen);
    torus_div2buf<<<nb, THREADS_PER_BLOCK>>>();
    hipDeviceSynchronize();

    fftcuda(torus_cpu.fftbuf);
    PoissonMain<<<nb, THREADS_PER_BLOCK>>>();
    hipDeviceSynchronize();

    ifftcuda(torus_cpu.fftbuf);
}

__global__ void StaggeredSharp_ker()
{
   // scale or "sharpen" the velocity fields (vx, vy, vz) 
   //of the simulation by dividing them by their respective grid spacings
   // (dx, dy, dz). This operation ensures that the velocity components are
   // normalized with respect to the grid resolution

    int i = check_limit(torus.plen);
    if(i<0) return; 
    torus.vx[i] /= torus.dx;
    torus.vy[i] /= torus.dy;
    torus.vz[i] /= turis.dz;

}

void Torus_StaggeredSharp()
{
    int nb = numblock(torus_cpu.plen);
    StaggeredSharp_ker<<<nb, THREADS_PER_BLOCK>>>();
    hipDeviceSynchronize();
}

__global__ void fftshift(hipfftDoubleComplex *data)
//This function is intended to perform an fftshift operation
// on a 3D array stored in data. In the context of Fourier transforms, 
//fftshift is often used to rearrange frequency-domain data so that
// the zero-frequency (DC) component is moved from the corner of the
// array to its center. This makes the frequency spectrum easier to interpret
// visually and numerically
{
  int xs = torus.resx / 2;
  int ys = torus.resy / 2;
  int zs = torus.resz / 2;
  int len = torus.plen;
  int x, y, z = 0;
  int j;

  /*if (len % 2 == 1){
    printf("Error: fftshift only supports even sized grid!\n");
    return;
  }*/

  int i = check_limit(len / 2);
  if(i<0) return;

  //for (int i=0; i<len/2; i++)
  //{
    hipfftDoubleComplex temp = data[i];
    getCoords(i, &x, &y, &z);
    x = (x + xs) % torus.resx;
    y = (y + ys) % torus.resy;
    z = (z + zs) % torus.resz;
    j = index3d(x, y, z);
    data[i] = data[j];
    data[j] = temp;
  //}
}

__global__ void ifftshift(hipfftDoubleComplex *data)
// Since we are only working with even-sized arrays
// ifftshift is equivalent with fftshift
{
  int xs = torus.resx / 2;
  int ys = torus.resy / 2;
  int zs = torus.resz / 2;
  int len = torus.resx * torus.resy * torus.resz;
  int x, y, z = 0;
  int j;

  if (len % 2 == 1){
    printf("Error: fftshift only supports even sized grid!\n");
    return;
  }

  for (int i=0; i<len/2; i++)
  {
    hipfftDoubleComplex temp = data[i];
    getCoords(i, &x, &y, &z);
    x = (x + xs) % torus.resx;
    y = (y + ys) % torus.resy;
    z = (z + zs) % torus.resz;
    j = index3d(x, y, z);
    data[i] = data[j];
    data[j] = temp;
  }
}
