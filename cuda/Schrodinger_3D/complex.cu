#include "hip/hip_runtime.h"
__device__ __inline__ 
void complexProd(hipDoubleComplex* a, double b)
//x, y risp. parte reale e immaginaria come definite nella libreria hip/hip_complex.h
{
    a -> x *= b;
    a -> y *= b;
}

__device__ __inline__ hipDoubleComplex exp_complex(hipDoubleComplex a)
{
    hipDoubleComplex res;
    res.x = exp(a.x)*cos(a.y);
    res.y = exp(a.x)*sin(a.y);
    return res;
}

__device__ __inline__ void complexDiv(hipDoubleComplex* n, double d)
{
    n -> x /= d;
    n -> y /= d;
}