#include "hip/hip_runtime.h"
// svdcuda.cu
#include "hip/hip_runtime.h"
#include ""

#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <cassert>
#include <math.h>
#include <algorithm>  // for std::min

#include <hipsolver.h>
#include <hip/hip_runtime_api.h>

#include "Utilities.cuh"  // Provides gpuErrchk, cusolveSafeCall, and iDivUp

/**
 * Compute the Singular Value Decomposition (SVD) of a matrix using cuSOLVER.
 *
 * The input matrix h_A is assumed to be stored in column-major order with dimensions:
 *     Nrows x Ncols   (with Nrows >= Ncols)
 *
 * The SVD computed is: h_A = U * S * V^T, where:
 *   - U is an (Nrows x Nrows) matrix (left singular vectors)
 *   - S is a vector of length min(Nrows, Ncols) (singular values)
 *   - V is an (Ncols x Ncols) matrix (right singular vectors)
 *
 * @param h_A   Pointer to the input host matrix (size: Nrows * Ncols)
 * @param Nrows Number of rows in the matrix.
 * @param Ncols Number of columns in the matrix.
 * @param h_S   Pointer to the output array for singular values (size: min(Nrows, Ncols)).
 * @param h_U   Pointer to the output array for U (size: Nrows * Nrows).
 * @param h_V   Pointer to the output array for V (size: Ncols * Ncols).
 */
void svdCudas(const double* h_A, int Nrows, int Ncols,
             double* h_S, double* h_U, double* h_V)
{
    // Ensure Nrows >= Ncols as required by gesvd.
    int minDim = std::min(Nrows, Ncols);

    // --- Create cuSOLVER handle.
    hipsolverHandle_t solver_handle;
    cusolveSafeCall(hipsolverDnCreate(&solver_handle));

    // --- Allocate device memory for matrix A and copy h_A into device memory.
    double* d_A = nullptr;
    gpuErrchk(hipMalloc(&d_A, Nrows * Ncols * sizeof(double)));
    gpuErrchk(hipMemcpy(d_A, h_A, Nrows * Ncols * sizeof(double), hipMemcpyHostToDevice));

    // --- Allocate device memory for S, U, and V.
    double *d_S = nullptr, *d_U = nullptr, *d_V = nullptr;
    gpuErrchk(hipMalloc(&d_S, Nrows*Ncols * sizeof(double)));
    gpuErrchk(hipMalloc(&d_U, Nrows * Nrows * sizeof(double)));
    gpuErrchk(hipMalloc(&d_V, Ncols * Ncols * sizeof(double)));

    // --- Allocate device memory for info output.
    int* devInfo = nullptr;
    gpuErrchk(hipMalloc(&devInfo, sizeof(int)));

    // --- Query working space for SVD.
    int work_size = 0;
    cusolveSafeCall(hipsolverDnDgesvd_bufferSize(solver_handle, Nrows, Ncols, &work_size));
    double* d_work = nullptr;
    gpuErrchk(hipMalloc(&d_work, work_size * sizeof(double)));

    // --- Execute SVD.
    // 'A' for jobu and jobvt indicates that all columns of U and V^T are computed.
    cusolveSafeCall(hipsolverDnDgesvd(solver_handle, 'A', 'A',
                                     Nrows, Ncols, d_A, Nrows,
                                     d_S, d_U, Nrows, d_V, Ncols,
                                     d_work, work_size, NULL, devInfo));

    // --- Check execution status.
    int devInfo_h = 0;
    gpuErrchk(hipMemcpy(&devInfo_h, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    if (devInfo_h != 0) {
        std::cerr << "SVD failed, devInfo = " << devInfo_h << std::endl;
        exit(EXIT_FAILURE);
    }

    // --- Copy results from device to host.
    gpuErrchk(hipMemcpy(h_S, d_S, minDim * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_U, d_U, Nrows * Nrows * sizeof(double), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(h_V, d_V, Ncols * Ncols * sizeof(double), hipMemcpyDeviceToHost));

    // --- Free device memory and destroy the solver handle.
    gpuErrchk(hipFree(d_A));
    gpuErrchk(hipFree(d_S));
    gpuErrchk(hipFree(d_U));
    gpuErrchk(hipFree(d_V));
    gpuErrchk(hipFree(devInfo));
    gpuErrchk(hipFree(d_work));
    hipsolverDnDestroy(solver_handle);
}
