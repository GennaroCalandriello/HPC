#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <string.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <sstream>
#include <string>
#include <vector>
#include <algorithm>
#include <hipsolver.h>
#include <hipblas.h>
#include "scalar.h"  // Defines DIM and MAX_FRAMES

using namespace std;
enum class Layout { RowMajor, ColMajor };

// Global flags to control saving behavior
bool savesigma   = true;  // Save singular values if true
int  pod_to_save = 10;    // Number of POD modes (columns) to save

// Macros to check CUDA and cuSOLVER errors
#define CHECK_CUDA(call) {                                  \
    hipError_t err = (call);                               \
    if (err != hipSuccess) {                               \
        fprintf(stderr, "CUDA error %s at line %d: %s\n",    \
                __FILE__, __LINE__, hipGetErrorString(err));\
        exit(EXIT_FAILURE);                                 \
    }                                                       \
}

#define CHECK_CUSOLVER(call) {                                \
    hipsolverStatus_t err = (call);                            \
    if (err != HIPSOLVER_STATUS_SUCCESS) {                     \
        fprintf(stderr, "cuSolver error %s at line %d: %d\n",  \
                __FILE__, __LINE__, err);                     \
        exit(EXIT_FAILURE);                                 \
    }                                                       \
}



void loadCsv(const std::string& filename,
             std::vector<double>& matrix,
             int& rows,
             int& cols,
             Layout layout = Layout::RowMajor) {
    std::ifstream file(filename);
    if (!file.is_open())
        throw std::runtime_error("Cannot open file: " + filename);

    std::vector<std::vector<double>> tempData;
    std::string line;

    while (std::getline(file, line)) {
        std::stringstream ss(line);
        std::string val;
        std::vector<double> row;

        while (std::getline(ss, val, ',')) {
            row.push_back(std::stod(val));
        }

        if (!tempData.empty() && row.size() != tempData[0].size())
            throw std::runtime_error("Inconsistent number of columns in CSV");

        tempData.push_back(row);
    }

    rows = tempData.size();
    cols = tempData[0].size();
    matrix.resize(rows * cols);

    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            if (layout == Layout::RowMajor)
                matrix[i * cols + j] = tempData[i][j];
            else // ColMajor
                matrix[j * rows + i] = tempData[i][j];
        }
    }
}

void printMatrix(const std::vector<double>& matrix, int rows, int cols, Layout layout = Layout::RowMajor) {
    int nrow = 100;
    int ncol =5;
    for (int i = 0; i < nrow; ++i) {
        for (int j = 0; j < ncol; ++j) {
            double val;
            if (layout == Layout::RowMajor)
                val = matrix[i * cols + j];
            else // ColMajor
                val = matrix[j * rows + i];

            std::cout << val << "\t";
        }
        std::cout << "\n";
    }
}

void saveMatrix(const std::string &filename,
                   const std::vector<double> &mat,
                   int N, int M,
                   Layout order =Layout::RowMajor)
{
    std::ofstream outFile(filename);
    if (!outFile.is_open()) {
        std::cerr << "Error opening file for writing: " << filename << std::endl;
        return;
    }

    // Write matrix elements to file.
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < M; j++) {
            double value = 0.0;
            if (order == Layout::ColMajor) {
                // In column-major order, element (i, j) is at index i + j*N.
                value = mat[i + j * N];
            } else {
                // In row-major order, element (i, j) is at index i*M + j.
                value = mat[i * M + j];
            }
            outFile << value;
            if (j < M - 1) {
                outFile << ",";
            }
        }
        outFile << "\n";
    }
    outFile.close();
}


void svdCudas(double* h_A, const int N, const int M,
              double* h_S, double* h_U, double* h_V)
{
    hipsolverHandle_t cusolverH = NULL;
    hipblasHandle_t cublasH = NULL;
    hipblasStatus_t cublas_status = HIPBLAS_STATUS_SUCCESS;
    hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    hipError_t cudaStat5 = hipSuccess;
    hipError_t cudaStat6 = hipSuccess;

    double *d_A = NULL,
           *d_S = NULL,
           *d_U = NULL,
           *d_VT = NULL;
    int *devInfo = NULL;
    double *d_work = NULL;
    double *d_rwork = NULL;
    double *d_W = NULL;  // W = S*VT

    int lwork = 0;
    int info_gpu = 0;
    const double h_one = 1;
    const double h_minus_one = -1;
    // step 1: create cusolverDn/cublas handle
    cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);

    cublas_status = hipblasCreate(&cublasH);
    assert(HIPBLAS_STATUS_SUCCESS == cublas_status);

    // step 2: copy A and B to device
    cudaStat1 = hipMalloc((void**)&d_A, sizeof(double) * N * M);
    cudaStat2 = hipMalloc((void**)&d_S, sizeof(double) * M);
    cudaStat3 = hipMalloc((void**)&d_U, sizeof(double) * N * N);
    cudaStat4 = hipMalloc((void**)&d_VT, sizeof(double) * M * M);
    cudaStat5 = hipMalloc((void**)&devInfo, sizeof(int));
    cudaStat6 = hipMalloc((void**)&d_W, sizeof(double) * N * M);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
    assert(hipSuccess == cudaStat5);
    assert(hipSuccess == cudaStat6);

    cudaStat1 = hipMemcpy(d_A, h_A, sizeof(double) * N * M, hipMemcpyHostToDevice);
    assert(hipSuccess == cudaStat1);

    //step 3 query working space of SVD
    cusolver_status = hipsolverDnDgesvd_bufferSize(cusolverH, N, M, &lwork);
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);

    cudaStat1=hipMalloc((void**)&d_work, sizeof(double)*lwork);
    assert(hipSuccess == cudaStat1);

    //step 4 compute svd
    signed char jobu = 'A'; // all m columns of U
    signed char jobvt = 'A'; // all n columns of VT
    //ATTENZIONE QUA, cusolver_status = hipsolverDnDgesvd (
        // cusolverH,
        // jobu,
        // jobvt,
        // N,
        // M,
        // d_A,
        // N,
        // d_S,
        // d_U,
        // N,  // ldu
        // d_VT,
        // N, // ldvt, GNOGNOGNOGNOGNOGNOGNO!!!
        // d_work,
        // lwork,
        // d_rwork,
        // devInfo);
    cusolver_status = hipsolverDnDgesvd(cusolverH, jobu, jobvt, N, M,
                                       d_A, N, d_S, d_U, N, d_VT, M,
                                       d_work, lwork, d_rwork,
                                       devInfo);
    cudaStat1 = hipDeviceSynchronize();
    assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
    assert (hipSuccess == cudaStat1);

    cudaStat1 = hipMemcpy(h_S, d_S, sizeof(double) * M, hipMemcpyDeviceToHost);
    cudaStat2 = hipMemcpy(h_U, d_U, sizeof(double) * N * N, hipMemcpyDeviceToHost);
    cudaStat3 = hipMemcpy(h_V, d_VT, sizeof(double) * M * M, hipMemcpyDeviceToHost);
    cudaStat4 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);

    //info GPU
    printf("after gesvd: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);
    printf("=====\n");

    printf("S = (matlab base-1)\n");
    printf("=====\n");

    printf("U = (matlab base-1)\n");
    printf("=====\n");

    printf("VT = (matlab base-1)\n");
    printf("=====\n");

    //free resources
    // free resources
    if (d_A    ) hipFree(d_A);
    if (d_S    ) hipFree(d_S);
    if (d_U    ) hipFree(d_U);
    if (d_VT   ) hipFree(d_VT);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);
    if (d_rwork) hipFree(d_rwork);
    if (d_W    ) hipFree(d_W);

    if (cublasH ) hipblasDestroy(cublasH);
    if (cusolverH) hipsolverDnDestroy(cusolverH);

    hipDeviceReset();

}


int main(){
    int N = 2*DIM*DIM;
    int M = MAX_FRAMES; // Number of frames (columns)
    string filename = "snapshots.csv";
    
    //    std::cout << "Row-major matrix:\n";
    // printMatrix(mat, rows, cols, Layout::RowMajor);
    vector<double> data;
    loadCsv(filename,data, N, M, Layout::RowMajor);
    // printMatrix(data, N, M, Layout::RowMajor);
    // cout << "------------------------\n";
    // printMatrix(data, N, M, Layout::ColMajor);
    int mn = std::min(N, M);
    vector<double> S(mn, 0.0);
    vector<double> U(N * N, 0.0);
    vector<double> Vt(M * M, 0.0);
    cout <<"Qua ngi arrivi mannaggiacristo e la madonna\n";
    //reconstruct the Sigma matrix
    


    // Perform SVD
    svdCudas(data.data(), N, M, S.data(), U.data(), Vt.data());
    vector<double> Sigma(N*M, 0.0);
    for (int i = 0; i < mn; ++i) {
        Sigma[i * N + i] = S[i];
        cout << S[i] << "\t";
    }
    //print each matrix
    // printMatrix(Sigma, N, M, Layout::RowMajor);
    // cout << "------------------------\n";
    // printMatrix(U, N, N, Layout::RowMajor);
    // cout << "------------------------\n";
    // printMatrix(Vt, M, M, Layout::RowMajor);
    // cout << "------------------------\n";
    //save
    saveMatrix("Sigma.txt", Sigma, N, M, Layout::RowMajor);
    saveMatrix("Vt.txt", Vt, M, M, Layout::RowMajor);
    saveMatrix("U.txt", U, N, N, Layout::RowMajor);

    return 0;
}